#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>

#include "helpFunction.h"
#include "constant.h"
#include "ising.h"
#include "in_out_functions.h"

__global__ void metropolisKernelTest(double* dev_H, double* dev_DelH, int* dev_DelH_sign, int* dev_Y, int* dev_Selected_index, int dev_lenY, double* dev_E, int* dev_bestSpinModel, double* best_energy, int exchange_attempts, double T)
{
    int tid = threadIdx.x;// +blockIdx.x * blockDim.x;
    //int i = threadIdx.x;
    hiprandState state;

    //int* dev_Selected_index
    //extern __shared__ int dev_Selected_index[];
    if (tid < dev_lenY) {
        for (int step = 1; step < exchange_attempts; step++) {
            //compute Delata energy
            double deltaE = -1 * (1 - dev_Y[tid]) * dev_H[tid];
            
            // Make decision that a bit flip can be accepted
            hiprand_init(0, tid, 0, &state);
            //hiprand_init(seed, tid, 0, &state);
            //double pr = hiprand_uniform_double(&state)
            
            if ((deltaE < 0) || (hiprand_uniform_double(&state) < exp(-deltaE / T))) {
                dev_Selected_index[tid] = tid;
            }
            /*else {
                dev_Selected_index[tid] = -1;
            }*/
            
            // select which bit accepted
            for (int s = dev_lenY; s > 0; s >>= 1)
            {
                if (tid < s)
                {
                    if (dev_Selected_index[tid] != -1 && dev_Selected_index[tid + s] != -1)
                    {
                        // Generate a random integer (0 or 1)

                        if ((hiprand(&state) % 2) == 1)
                            dev_Selected_index[tid] = tid + s;
                    }
                    else if (dev_Selected_index[tid] == -1 && dev_Selected_index[tid + s] != -1) {
                        dev_Selected_index[tid] = tid + s;
                    }
                }
            }
            __syncthreads();
            // based on flipped bit do some calculation
            if (tid == dev_Selected_index[0]) {
                dev_Y[tid] = 1 - dev_Y[tid];
                dev_E[step] = dev_E[step - 1] + deltaE;
                if (dev_E[step] < *best_energy) {
                    *best_energy = dev_E[step];
                    dev_bestSpinModel[tid] = dev_Y[tid];
                }
            }
            __syncthreads();


            dev_H[tid] = dev_H[tid] + dev_DelH[tid + dev_Selected_index[0] * dev_lenY];
            //dev_H[tid] = dev_H[tid] + dev_DelH[tid][dev_Selected_index[0]] * dev_DelH_sign[tid];


            if (tid == dev_Selected_index[0]) {
                for (int i = 0; i < dev_lenY; i++) {
                    dev_DelH[tid + dev_Selected_index[0] * dev_lenY] *= -1;
                }
            }


            //dev_bestSpinModel[i] *= (1 - dev_Y[i]);
            //dev_E[counter] = exchange_attemps - counter;
        }
    }
}

void checkErrorCuda(hipError_t cudaStatus, string message) {
    if (cudaStatus != hipSuccess) {
        cout << message << " : " << hipGetErrorString(cudaStatus) << endl;
        //fprintf(stderr, "%s : %s \n",  message, hipGetErrorString(cudaStatus));
    }
}

hipError_t prepareMetropolisKernel(double* H, double* DelHGpu, int* DelH_sign, double* WGpu, double* B, int* Y, int lenY, double* M, double* E, double T, int step, int exchange_attempts, double& bestEnergy, int* bestSpinModel, int replica) {
    
    double* dev_H = 0;
    double* dev_DelH = 0;
    double* dev_W = 0;
    double* dev_E = 0;

    int* dev_bestSpinModel = 0;
    int* dev_Y = 0;
    int* dev_Selected_index=0;
    int* dev_lenY = 0;
    int* dev_DelH_sign = 0;
    double* dev_bestenergy = 0;

    //double* dev_oldE;
    //double* dev_B = 0;
    
    //double* dev_M = 0;


    hipError_t cudaStatus;
    cout<<"replica: "<< replica << " bestEnergy: " << bestEnergy << "\t\t\t\t\t In prepareMetropolisKernel  first lines \n";

    
    

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    checkErrorCuda(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    
    // Allocate GPU buffers for inputs and outputs.
    cudaStatus = hipMalloc((void**)&dev_H, lenY * sizeof(double));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_H");
    cudaStatus = hipMalloc((void**)&dev_DelH, lenY * lenY * sizeof(double));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_DelH");
    cudaStatus = hipMalloc((void**)&dev_DelH_sign, lenY * sizeof(int));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_DelH_sign");
    cudaStatus = hipMalloc((void**)&dev_W, lenY * lenY * sizeof(double));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_W");
    //hipMalloc((void**)&dev_B, lenY * sizeof(double));
    cudaStatus = hipMalloc((void**)&dev_Y, lenY * sizeof(int));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_Y");
    cudaStatus = hipMalloc((void**)&dev_Selected_index, lenY * sizeof(int));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_Selected_index");
    cudaStatus = hipMalloc((void**)&dev_E, (exchange_attempts + 1) * sizeof(double));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_E");
    cudaStatus = hipMalloc((void**)&dev_bestSpinModel, lenY * sizeof(int));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_bestSpinModel");
    //hipMalloc((void**)&dev_Flag, lenY * sizeof(int));
    cudaStatus = hipMalloc((void**)&dev_bestenergy, sizeof(double));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_bestenergy");
    cudaStatus = hipMalloc((void**)&dev_lenY, sizeof(int));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_lenY");
        


        

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_H, H, lenY * sizeof(double), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_H");
    cudaStatus = hipMemcpy(dev_DelH, DelHGpu, lenY * lenY * sizeof(double), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_DelH");
    cudaStatus = hipMemcpy(dev_W, WGpu, lenY * lenY * sizeof(double), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_W");
    //hipMemcpy(dev_B, B, lenY * sizeof(double), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_Y, Y, lenY * sizeof(int), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_Y");
    cudaStatus = hipMemcpy(dev_bestSpinModel, bestSpinModel, lenY * sizeof(int), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_bestSpinModel");
    cudaStatus = hipMemcpy(dev_bestenergy, &bestEnergy, sizeof(double), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_bestenergy");
    //hipMemset(dev_bestenergy, bestEnergy, sizeof(double));
    cudaStatus = hipMemcpy(dev_E, E + step - 1, sizeof(double), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_E");
    cudaStatus = hipMemset(dev_lenY, lenY, sizeof(int));
    checkErrorCuda(cudaStatus, "hipMemset failed! dev_lenY");

    cudaStatus = hipMemset(dev_Selected_index, -1, lenY * sizeof(int));
    checkErrorCuda(cudaStatus, "hipMemset failed! dev_Selected_index");
    
    
    cudaStatus = hipMemcpy(dev_DelH_sign, DelH_sign, lenY * sizeof(int), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_DelH_sign");
    //hipMemset(dev_E, E[step-1], sizeof(double));
    //hipMemset(dev_oldE, E[step-1], sizeof(double));

    
    cout << "\t\tBefore metropolisKernelTest" << endl;
    
    // Launch a kernel on the GPU with one thread for each element.
    metropolisKernelTest <<<1, lenY >>> (dev_H, dev_DelH, dev_DelH_sign, dev_Y, dev_Selected_index, lenY, dev_E, dev_bestSpinModel, dev_bestenergy, exchange_attempts, T);

    cudaStatus = hipGetLastError();
    checkErrorCuda(cudaStatus, "prepareMetropolisKernel launch failed : !");
    cout << "\t\tafter metropolisKernelTest" << endl;

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "prepareMetropolisKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching metropolisKernel!\n", cudaStatus);
        cout << hipGetErrorString(cudaStatus) << endl;;
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.

    hipMemcpy(H, dev_H, lenY * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(DelHGpu, dev_DelH, lenY * lenY * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Y, dev_Y, lenY * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(bestSpinModel, dev_bestSpinModel, lenY * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&bestEnergy, dev_bestenergy, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(E + step, dev_E, exchange_attempts * sizeof(double), hipMemcpyDeviceToHost);
    //hipMemcpy(M + step, dev_M, exchange_attempts * sizeof(double), hipMemcpyDeviceToHost);

    cudaStatus = hipMemcpy(DelH_sign, dev_DelH_sign, lenY * sizeof(int), hipMemcpyDeviceToHost);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_DelH_sign -> DelH_sign");

    cout << " \t Best Energy: " << bestEnergy << endl;

Error:
    hipFree(dev_H);
    hipFree(dev_DelH);
    hipFree(dev_DelH_sign);
    hipFree(dev_W);
    //hipFree(dev_B);
    hipFree(dev_Y);
    hipFree(dev_E);
    hipFree(dev_Selected_index);
    hipFree(dev_bestSpinModel);
    hipFree(dev_bestenergy);


    return cudaStatus;


}

  
hipError_t prepareMetropolisKernel2(double* H, double* DelHGpu, double* WGpu, double* B, int* Y, int lenY, double* M, double* E, double T, int step, int exchange_attempts, double& bestEnergy, int* bestSpinModel) {
    
       
    double* dev_H = 0;
    double* dev_DelH = 0;
    int* dev_DelH_sign;
    double* dev_W = 0;
    //double* dev_B = 0;
    int* dev_Y = 0;
    double* dev_bestenergy = 0;
    //double* dev_oldE;


    //int* dev_Flag;

    double* dev_E = 0;
    //double* dev_M = 0;
    int* dev_bestSpinModel=0;
    
    hipError_t cudaStatus;


    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for inputs and outputs.
    hipMalloc((void**)&dev_H, lenY * sizeof(double));
    hipMalloc((void**)&dev_DelH, lenY * lenY * sizeof(double));
    hipMalloc((void**)&dev_DelH_sign, lenY * sizeof(int));
    hipMalloc((void**)&dev_W, lenY * lenY * sizeof(double));
    //hipMalloc((void**)&dev_B, lenY * sizeof(double));
    hipMalloc((void**)&dev_Y, lenY * sizeof(int));
    hipMalloc((void**)&dev_E, (exchange_attempts+1) * sizeof(double));
    hipMalloc((void**)&dev_bestSpinModel, lenY * sizeof(int));
    //hipMalloc((void**)&dev_Flag, lenY * sizeof(int));
    hipMalloc((void**)&dev_bestenergy, sizeof(double));
    


    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "prepareMetropolisKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    goto Error;

    
    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_H, H, lenY * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_DelH, DelHGpu, lenY * lenY * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_W, WGpu, lenY * lenY * sizeof(double), hipMemcpyHostToDevice);
    //hipMemcpy(dev_B, B, lenY * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_Y, Y, lenY * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_bestSpinModel, bestSpinModel, lenY * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_bestenergy, &bestEnergy, sizeof(double), hipMemcpyHostToDevice);
    //hipMemset(dev_bestenergy, bestEnergy, sizeof(double));
    hipMemcpy(dev_E, E+step - 1, sizeof(double), hipMemcpyHostToDevice);
    //hipMemset(dev_E, E[step-1], sizeof(double));
    //hipMemset(dev_oldE, E[step-1], sizeof(double));

    cout << "before metropolisKernelTest" << endl;
    // Launch a kernel on the GPU with one thread for each element.
    //metropolisKernelTest <<<1, lenY >>> (dev_H, dev_DelH, dev_DelH_sign, dev_Y, lenY, dev_E, dev_bestSpinModel, dev_bestenergy, exchange_attempts, T);
   
    
    cout << "after metropolisKernelTest" << endl;

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "prepareMetropolisKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching metropolisKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.

    hipMemcpy(H, dev_H, lenY * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(DelHGpu, dev_DelH, lenY * lenY * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Y, dev_Y, lenY * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(bestSpinModel, dev_bestSpinModel, lenY * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&bestEnergy, dev_bestenergy, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(E + step, dev_E, exchange_attempts * sizeof(double), hipMemcpyDeviceToHost);
    //hipMemcpy(M + step, dev_M, exchange_attempts * sizeof(double), hipMemcpyDeviceToHost);
       
    cout << bestEnergy << endl;

Error:
    hipFree(dev_H);
    hipFree(dev_DelH);
    hipFree(dev_DelH_sign);
    hipFree(dev_W);
    //hipFree(dev_B);
    hipFree(dev_Y);
    hipFree(dev_E);
    //hipFree(dev_Flag);
    hipFree(dev_bestSpinModel);
    hipFree(dev_bestenergy);


    return cudaStatus;


}

/* ***************************************************************** */


__global__ void metropolisKernel(double* dev_H, double* dev_DelH, double* dev_W, double* dev_B, int* dev_Y, double* dev_E, double* dev_M, int* dev_bestSpinModel)
{
    //int i = threadIdx.x;
    
}





// One step in metropolis algorithm
double metropolis(int ExecuteMode, double** W, double* B, double* H, double** DelH, int* X, int lenX, double OldE, double T, int step, int replica) {
    double E = 0;
    int i = rand() % lenX;

    double deltaE;
    if (ExecuteMode == IsingMode)    deltaE = deltaEnergyIsing(ExecuteMode, W, B, X, lenX, i);
    else if (ExecuteMode == QUBOMode) deltaE = deltaEnergyQUBO(W, X, i, H);

    if ((deltaE < 0) || ((rand() / static_cast<double>(RAND_MAX)) < exp(-deltaE / T))) {
        if (ExecuteMode == IsingMode) {
            X[i] *= -1;
            E = energy(W, B, X, lenX);
        }
        else if (ExecuteMode == QUBOMode) {
            X[i] = 1 - X[i];
            E = Energy_based_Delta(OldE, deltaE);
            updateH(H, DelH, lenX, i);
            update_delta_H(DelH, i, lenX);
        }
        return E;
    }
    return OldE;
}



/* Execute the optimization function based on replica exchange MCMC
* ExecuteMode : Ising or QUBO
* W: interconnection of spins
* B: bias
* Y: spins 2D array sizeof (num_replicas \times lenY), lenY: number of spins
* M: record of Magnet in each iteration per replica
* E: record of Energy in each iteration per replica
*/
void ising(int ExecuteMode, double** W, double* B, int** Y, int lenY, double** M, double** E, double T, int num_replicas, int numberOfIteration, int exchange_attempts, int* bestSpinModel, double minTemp, double maxTemp) {

    double bestEnergy;
    double* Temperature = new double[num_replicas];
    double** H = ComputeH_forAllReplica(num_replicas, W, B, Y, lenY);
    double*** DelH = ComputeDelH_forAllReplica(num_replicas, W, Y, lenY);
    int** DelH_sign = Declare2D_ArrayInt(num_replicas, lenY);
    fill2DarrayInt(DelH_sign, 1, num_replicas, lenY);
    double** DelHGpu;
    double* WGpu;
    
    

    //testHamiltonianPreparation(W, B, lenY);
    //int tempValue;
    //cin >> tempValue;

    //initialize the bestEnergy, Temperature array range, Energy (E), Magnet (M), and bestSpinModel
    if (num_replicas != 1) {
        for (int r = 0; r < num_replicas; r++) {
            Temperature[r] = minTemp + r * (maxTemp - minTemp) / (num_replicas - 1);
            cout << "Temperature: " << Temperature[r] << endl;
            E[r][0] = energy(W, B, Y[r], lenY);

            M[r][0] = magnetization(Y[r], lenY);
            if (r == 0 || bestEnergy > E[r][0]) {
                bestEnergy = E[r][0];
                memcpy(bestSpinModel, Y[r], sizeof(int) * lenY);
            }
        }
    }
    else {
        Temperature[0] = minTemp;
        cout << "Temperature: " << Temperature[0] << endl;
        E[0][0] = energy(W, B, Y[0], lenY);
        M[0][0] = magnetization(Y[0], lenY);
        bestEnergy = E[0][0];
        memcpy(bestSpinModel, Y[0], sizeof(int) * lenY);
    }

    if (ExecuteMode == QUBOGPU) {
        DelHGpu = convertDelHtoGpuDelH(DelH, num_replicas, lenY);
        WGpu = convert2Dto1D(W, lenY, lenY);
    }
    

    // Preperation of replica exchange parameters
    int exchangeFlag = 0;   // Flag to enable the exchange between neighbour replicas
    // Perform the Metropolis function numberOfIteration times for each replica 
    for (int step = 1; step < numberOfIteration; step++) {
        //cout << "step: " << step << endl;
        cout << "before prepareMetropolisKernel calling/ step: " << step << endl;
        for (int r = 0; r < num_replicas; r++) {
            T = Temperature[r];
            double previousE = E[r][step - 1];
            //for (int spin = 0; spin < lenY/25; spin++) 
            
            if (ExecuteMode == QUBOGPU) {                
                printX(Y[r], lenY);
                printH(H[r], lenY);
                print2D_arr_double(DelH[r], lenY, lenY);
                prepareMetropolisKernel(H[r], DelHGpu[r], DelH_sign[r], WGpu, B, Y[r], lenY, M[r], E[r], T, step, exchange_attempts, bestEnergy, bestSpinModel, r);
                

                
            }
            else {
                previousE = metropolis(ExecuteMode, W, B, H[r], DelH[r], Y[r], lenY, previousE, T, step, r);
                E[r][step] = previousE;
                M[r][step] = magnetization(Y[r], lenY);
                if (bestEnergy > E[r][step]) {
                    memcpy(bestSpinModel, Y[r], sizeof(int) * lenY);
                    bestEnergy = E[r][step];
                }
            }            
        }
        // Replica exchange attempts
        if (ExecuteMode != QUBOGPU && step % exchange_attempts == 0) {
            replicaExchange(Temperature, num_replicas, Y, M, E, step, H, DelH, exchangeFlag);
        }
        else if (ExecuteMode == QUBOGPU) {
            step = step + exchange_attempts-1;
            cout << "after prepareMetropolisKernel calling/ step: " << step << endl;
        }

    }
}

/**********************************************************************************************/





int main()
{
    


    int L = 0;                      // Number of spins for each replica
    int Lsqrt = 0;                  /* This parameter is just used for graphical representation of 2-D Ising model
                                     and it is not important for other cases Lsqrt * Lsqrt = L         */
    double T = 0.0;                 // Temperature ---> when 
    int num_replicas = 1;           // Number of replica in the replica exchange MCMC
    double minTemp = 0;             // Min temperature of replica exchange MCMC
    double maxTemp = 0;             // Max temperature of replica exchange MCMC
    int numberOfIteration = 1;      // Number of Iteration in for MCMC
    int exchange_attempts = 0;      // After how many iteration, an exchange should be applied

    /*      E = -\sum_{i,j} A_{i,j} s_i s_j - \sum_i B_i s_i   */
    string Afile = "";              //      file path for A matrix in Ising (QUBO) model 
    string Bfile = "";              //      file path for Bias (B) in Ising (QUBO) model 
    string outputPath = "";          //     The path to the directory to save the output 
    int ExecuteMode = QUBOMode;     //      Execution mode: IsingMode or QUBOMode

    /*      Read the setting file and initialize the parameters    */
    readSetting(L, Lsqrt, Afile, Bfile, outputPath, ExecuteMode, num_replicas, numberOfIteration, exchange_attempts, minTemp, maxTemp);

    //cout << "L: " << L << " Lsqrt: " << Lsqrt << " Afile: " << Afile << " Bfile: " << Bfile << " ExecuteMode: " << ExecuteMode << " num_replicas: " << num_replicas << " numberOfIteration: " << numberOfIteration << " exchange_attempts: " << exchange_attempts << endl;
    //cout << "minTemp: " << minTemp << " maxTemp: " << maxTemp << endl;


    // Create the output folder
    createFolder(outputPath);

    //Initialize the spins for each replica
    int** X = createVector(ExecuteMode, L, num_replicas);
    int* bestSpinModel = new int[L];            // Best found solution

    double** M = new double* [num_replicas];     //  Magnetization for different replica (This parameter is usefull for Ising model)
    double** E = new double* [num_replicas];     //  Energy for different replica
    for (int r = 0; r < num_replicas; r++) {
        M[r] = new double[numberOfIteration];
        E[r] = new double[numberOfIteration];
    }

    //  Initialize the bias    
    double* B;
    B = initB(L, ReadDataFromFile, Bfile);

    //Initalize the interconnection of spins
    double** A;
    A = initW(L, Lsqrt, ReadDataFromFile, Afile);
    //writeMatrixToFile(outputPath + "\\WInit.csv", A, L);

    // Log all the initial spin states
    writeSpinsInFile(num_replicas, X, L, Lsqrt, outputPath, "Initlattice");

    // Optimization Function
    ising(ExecuteMode, A, B, X, L, M, E, T, num_replicas, numberOfIteration, exchange_attempts, bestSpinModel, minTemp, maxTemp);

    //  Record Logs: Magnet, Energy, final spin states, and best spin model
    recordLogs(outputPath, M, E, numberOfIteration, num_replicas, L, Lsqrt, X, bestSpinModel);

    return 0;
}



/*****************************************************************************************/