#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>

#include "helpFunction.h"
#include "constant.h"
#include "ising.h"
#include "in_out_functions.h"


//Gpu kernel for metropolis function => Almost same as sequential version of metropolis just perform tasks in parallel.
__global__ void metropolisKernel(double* dev_H, double* dev_DelH, int* dev_DelH_sign, int* dev_Y, int* dev_Selected_index, int dev_lenY, double* dev_E, int* dev_bestSpinModel, double* best_energy, int exchange_attempts, double T)
{
    int tid = threadIdx.x;
    
    //extern __shared__ int sdata[];
    hiprandState state;

    if (tid < dev_lenY) {
        for (int step = 1; step < exchange_attempts; step++) {
            
            //compute Delata energy
            double deltaE = -1 * (1 - 2* dev_Y[tid]) * dev_H[tid];
            //printf("Thread %d: Local variable value = %f\n", threadIdx.x, deltaE);
            

            // Make decision that a bit flip can be accepted
            hiprand_init(clock64(), tid, clock64(), &state);
            
            if ((deltaE < 0) || (hiprand_uniform_double(&state) < exp(-deltaE / T))) {
                dev_Selected_index[tid] = tid;
            }
            else {
                dev_Selected_index[tid] = -1;
            }            
            //__syncthreads();

            // select which bit accepted            
            for (int s = blockDim.x / 2; s > 0; s >>= 1)
            {
                if (tid<s)
                {
                    if (dev_Selected_index[tid] != -1 && dev_Selected_index[tid + s] != -1)
                    {
                        // Generate a random integer (0 or 1)

                        if ((hiprand(&state) % 2) == 1)
                            dev_Selected_index[tid] = dev_Selected_index[tid + s];
                    }
                    else if (dev_Selected_index[tid] == -1 && dev_Selected_index[tid + s] != -1) {
                        dev_Selected_index[tid] = dev_Selected_index[tid + s];
                    }
                }
                __syncthreads();
            }
            
            __syncthreads();


            // based on the flipped bit j update parameters
            int j = dev_Selected_index[0];
            if (tid == j) {
                dev_Y[tid] = 1 - dev_Y[tid];
                dev_E[step] = dev_E[step - 1] + deltaE;
                if (dev_E[step] < *best_energy) {
                    *best_energy = dev_E[step];
                    dev_bestSpinModel[tid] = dev_Y[tid];
                }
                //printf("Thread %d: flipped bit = %d, En: %f -> %f (delE: %f) bestE: %f\n", tid, j, dev_E[step-1], dev_E[step], deltaE, *best_energy);
                //printf("\t\t step %d [%d, %d, %d, %d, %d]\n", step, dev_Y[0], dev_Y[1], dev_Y[2], dev_Y[3], dev_Y[4]);
            }
            __syncthreads();

            if (j != -1) {
                //Update H
                dev_H[tid] = dev_H[tid] + dev_DelH[tid + j * dev_lenY];                
            }
            else {
                // Log the Energy when there is not any bit to flip
                dev_E[step] = dev_E[step - 1];
            }
            
            // Update delta_H
            if (tid == j) {
                for (int i = 0; i < dev_lenY; i++) {
                    dev_DelH[i + j * dev_lenY] *= -1;                    
                }
            }            
        }
    }
}

//dev_H, dev_DelH, dev_DelH_sign, dev_Y, dev_Selected_index, lenY, dev_E, dev_bestSpinModel, dev_bestenergy, numberOfIteration
__global__ void full_mode_metropolisKernel(double* dev_H, double* dev_DelH, int* dev_DelH_sign, int* dev_Y, int* dev_Selected_index, const int select_index_size, double* dev_E, int* dev_bestSpinModel, double* dev_best_energy, int numberOfIteration, int exchange_attempts, double* dev_Temprature)
{   
    int blockId = blockIdx.x;
    int threadId = threadIdx.x;
    int tid = blockId * blockDim.x + threadId;
    int temprature_index = blockId; 
    int temp_index_direction = (blockId+1) % 2;
    bool stop_flag = false;

    int index_base = select_index_size* blockId;
    
    double bE = dev_best_energy[blockId];
    //for (int i = 0; i < 101; i++) {
    //    printf("dev_DelH[%d] = %lf\n", i, dev_DelH[i]);
    //}

    if (temp_index_direction==0) {
        temp_index_direction = -1;
        //printf("\tblockId %d temp_index_direction %d \n", blockId, temp_index_direction);
    }
    
    //if (threadId == 0) {
    //    printf("blockId %d, tempIndex %d, tempDirection %d \n", blockId, temprature_index, temp_index_direction);
    //}
    
    if (blockId == gridDim.x - 1 && blockId % 2 == 0) {
        stop_flag = true;
        temp_index_direction = -1;
        printf("blockId %d the stop_flag is on \n", blockId);
    }

    //extern __shared__ int sdata[];
    hiprandState state;

    //printf("\tblockId: %d \t threadId: %d \t index_base + threadId: %d\n", blockId, threadId, index_base + threadId);
     
    
    //if (tid < gridDim.x * blockDim.x) {

        for (int step = 1; step < numberOfIteration; step++) {
            
            //compute Delata energy
            double deltaE = -1 * (1 - 2 * dev_Y[tid]) * dev_H[tid];
            //printf("Thread %d: Local variable value = %f\n", threadIdx.x, deltaE);

            

            // Make decision that a bit flip can be accepted
            hiprand_init(clock64(), tid, clock64(), &state);

            if ((deltaE < 0) || (hiprand_uniform_double(&state) < exp(-deltaE / dev_Temprature[temprature_index]))) {
                dev_Selected_index[index_base+threadId] = threadId;
            }
            else {
                dev_Selected_index[index_base + threadId] = -1;
            }
            //__syncthreads();

            // select which bit accepted            
            for (int s = select_index_size / 2; s > 0; s >>= 1)
            {
                if (threadId < s)
                {
                    //printf("\tblockId: %d \t threadId: %d \t index_base + threadId: %d \t index_base + threadId+s: %d\n", blockId, threadId, index_base + threadId, index_base + threadId + s);
                    if (dev_Selected_index[index_base + threadId] != -1 && dev_Selected_index[index_base + threadId + s] != -1)
                    {
                        // Generate a random integer (0 or 1)

                        if ((hiprand(&state) & 1) == 1) //find the least significant bit
                            dev_Selected_index[index_base + threadId] = dev_Selected_index[index_base + threadId + s];
                    }
                    else if (dev_Selected_index[index_base + threadId] == -1 && dev_Selected_index[index_base + threadId + s] != -1) {
                        dev_Selected_index[index_base + threadId] = dev_Selected_index[index_base + threadId + s];
                    }
                }
                __syncthreads();
            }

            __syncthreads();

            //if (step == 1 && blockId <= 3)
            //    printf("\tblockId: %d \t threadId: %d \t dev_Selected_index %d \n", blockId, threadId, dev_Selected_index[index_base + threadId]);

            // based on the flipped bit j update parameters
            int j = dev_Selected_index[index_base];

            //if (threadId == j && step <= 2) {
                //printf("step: %d, tid: %d, blockId: %d, threadId: %d == %d :j, E: %f, deltaE: %f, NewE: %f \n", step, tid, blockId, threadId, j, dev_E[blockId * numberOfIteration + step-1], deltaE, dev_E[blockId * numberOfIteration + step - 1] + deltaE);
            //}

            if (threadId == j) {

                dev_Y[tid] = 1 - dev_Y[tid];
                dev_E[blockId * numberOfIteration + step] = dev_E[blockId * numberOfIteration + step - 1] + deltaE;
                if (dev_E[blockId * numberOfIteration + step] < dev_best_energy[blockId]) {
                    dev_best_energy[blockId] = dev_E[blockId * numberOfIteration + step];
                    dev_bestSpinModel[tid] = dev_Y[tid];
                }
                int Y_index=blockId* blockDim.x;
                if (blockId == 3 && DEBUGFLAG) {
                    printf("\tstep %d :: blockId: %d threadId %d: flipped bit = %d, En: %f -> %f (delE: %f) bestE: %f\n", step, blockId, threadId, j, dev_E[blockId * numberOfIteration + step - 1], dev_E[blockId * numberOfIteration + step], deltaE, dev_best_energy[blockId]);
                    printf("\t\t step %d :: blockId %d [%d, %d, %d, %d, %d]\n", step, blockId, dev_Y[Y_index + 0], dev_Y[Y_index + 1], dev_Y[Y_index + 2], dev_Y[Y_index + 3], dev_Y[Y_index + 4]);
                }
            }
            __syncthreads();

            if (j != -1) {
                //Update H                  ( dev_DelH : replica * lenY * lenY)
                if (step <=-1 && DEBUGFLAG)
                    printf("step: %d, [index: %d] , tid: %d, blockId: %d, threadId: %d, j: %d, dev_H: %f, dev_DelH: %f, new dev_H: %f \n", 
                    step,
                    blockId * blockDim.x * blockDim.x + threadId * blockDim.x + j,
                    tid,
                    blockId,
                    threadId,
                    j,
                    dev_H[tid],
                    dev_DelH[blockId * blockDim.x * blockDim.x + threadId * blockDim.x + j],
                    dev_H[tid]);

                double oldValue = dev_H[tid];

                dev_H[tid] = dev_H[tid] + dev_DelH[blockId * blockDim.x * blockDim.x + threadId * blockDim.x + j];
                //dev_H[tid] = dev_H[tid] + dev_DelH[tid + j * dev_lenY];
                //VectorDelH[r * num_replicas * lenY + i * lenY + i] = DelH[r][i][j];
                if (blockId == 3 && DEBUGFLAG) {
                    printf("Update dev_H: %lf to %lf \n", oldValue, dev_H[tid]);
                }

            }
            else {
                // Log the Energy when there is not any bit to flip
                dev_E[blockId * numberOfIteration + step] = dev_E[blockId * numberOfIteration + step - 1];
            }

            // Update delta_H
            if (threadId == j) {
                for (int i = 0; i < blockDim.x; i++) {
                    //double oldValue= dev_DelH[blockId * blockDim.x * blockDim.x + i * blockDim.x + j];
                    dev_DelH[blockId * blockDim.x * blockDim.x + i * blockDim.x + j] *= -1;
                    
                }
            }
            
            __syncthreads();

            if (step % exchange_attempts == 0) {
                if (stop_flag == false) {
                    
                    temprature_index += temp_index_direction;

                    if (temprature_index == 0 || temprature_index == gridDim.x-1) {
                        stop_flag = true;
                        temp_index_direction = temp_index_direction * -1;
                    }
                }
                else {
                    stop_flag = false;
                }                   
            }
            if (threadId == 0 && bE != dev_best_energy[blockId]) {
                printf("S %d bE in block %d is %lf \n", step, blockId, dev_best_energy[blockId]);
                bE = dev_best_energy[blockId];
            }
        }
    //}
}


// Show Error message
void checkErrorCuda(hipError_t cudaStatus, string message) {
    if (cudaStatus != hipSuccess) {
        cout << message << " : " << hipGetErrorString(cudaStatus) << endl;
        //fprintf(stderr, "%s : %s \n",  message, hipGetErrorString(cudaStatus));
    }
}

hipError_t allocateMemory(int lenY, int block_size, int exchange_attempts, double** dev_H, double** dev_DelH, double** dev_W, double** dev_B, double** dev_E, int** dev_bestSpinModel, int** dev_Y, int** dev_Selected_index, int** dev_lenY, int** dev_DelH_sign, double** dev_bestenergy) {

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system. (!For future!)
    cudaStatus = hipSetDevice(0);
    checkErrorCuda(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");

    // Allocate GPU buffers for inputs and outputs.
    cudaStatus = hipMalloc((void**)dev_H, lenY * sizeof(double));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_H");
    cudaStatus = hipMalloc((void**)dev_DelH, lenY * lenY * sizeof(double));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_DelH");
    cudaStatus = hipMalloc((void**)dev_DelH_sign, lenY * sizeof(int));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_DelH_sign");
    cudaStatus = hipMalloc((void**)dev_W, lenY * lenY * sizeof(double));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_W");


    hipMalloc((void**)&dev_B, lenY * sizeof(double));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_B");
    cudaStatus = hipMalloc((void**)dev_Y, lenY * sizeof(int));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_Y");
    cudaStatus = hipMalloc((void**)dev_Selected_index, block_size * sizeof(int));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_Selected_index");
    cudaStatus = hipMalloc((void**)dev_E, (exchange_attempts + 1) * sizeof(double));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_E");
    cudaStatus = hipMalloc((void**)dev_bestSpinModel, lenY * sizeof(int));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_bestSpinModel");
    //hipMalloc((void**)&dev_Flag, lenY * sizeof(int));
    cudaStatus = hipMalloc((void**)dev_bestenergy, sizeof(double));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_bestenergy");
    cudaStatus = hipMalloc((void**)dev_lenY, sizeof(int));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_lenY");

    return cudaStatus;

}

hipError_t allocateMemory_with_size(double** dev_H, double** dev_DelH, double** dev_W, double** dev_B, double** dev_E, int** dev_bestSpinModel, int** dev_Y, int** dev_Selected_index, int** dev_lenY, int** dev_DelH_sign, double** dev_bestenergy, double** dev_Temprature, int* sizeArray) {

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system. (!For future!)
    cudaStatus = hipSetDevice(0);
    checkErrorCuda(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");

    // Allocate GPU buffers for inputs and outputs.
    cudaStatus = hipMalloc((void**)dev_H, sizeArray[0] * sizeof(double));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_H");
    cudaStatus = hipMalloc((void**)dev_DelH, sizeArray[1] * sizeof(double));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_DelH");
    cudaStatus = hipMalloc((void**)dev_DelH_sign, sizeArray[2] * sizeof(int));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_DelH_sign");
    cudaStatus = hipMalloc((void**)dev_W, sizeArray[3] * sizeof(double));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_W");


    hipMalloc((void**)&dev_B, sizeArray[4] * sizeof(double));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_B");


    cudaStatus = hipMalloc((void**)dev_Y, sizeArray[5] * sizeof(int));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_Y");
    cudaStatus = hipMalloc((void**)dev_Selected_index, sizeArray[6] * sizeof(int));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_Selected_index");
    cudaStatus = hipMalloc((void**)dev_E, sizeArray[7] * sizeof(double));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_E");
    cudaStatus = hipMalloc((void**)dev_bestSpinModel, sizeArray[9] * sizeArray[8] * sizeof(int));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_bestSpinModel");
    //hipMalloc((void**)&dev_Flag, lenY * sizeof(int));
    cudaStatus = hipMalloc((void**)dev_bestenergy, sizeArray[9] * sizeof(double));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_bestenergy");

    cudaStatus = hipMalloc((void**)dev_Temprature, sizeArray[10] * sizeof(double));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_Temprature");

    cudaStatus = hipMalloc((void**)dev_lenY, sizeof(int));
    checkErrorCuda(cudaStatus, "hipMalloc failed! dev_lenY");

    return cudaStatus;

}

hipError_t copyMemoryFromHostToDevice_with_size(double* H, double* dev_H, double* DelH, double* dev_DelH, int* DelH_sign, int* dev_DelH_sign, double* WGpu, double* dev_W, int* Y, int* dev_Y, double* E, double* dev_E, double bestEnergy, double* dev_bestenergy, int* bestSpinModel, int* dev_bestSpinModel, int* dev_Selected_index, double* dev_Temprature, double* Temprature, int* sizeArray) {

    hipError_t cudaStatus;
    // Copy input vectors from host memory to GPU buffers.

    cudaStatus = hipMemcpy(dev_H, H, sizeArray[0] * sizeof(double), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_H");
    cudaStatus = hipMemcpy(dev_DelH, DelH, sizeArray[1] * sizeof(double), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_DelH");


    cudaStatus = hipMemcpy(dev_W, WGpu, sizeArray[3] * sizeof(double), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_W");

  
    cudaStatus = hipMemcpy(dev_Y, Y, sizeArray[5] * sizeof(int), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_Y");
    cudaStatus = hipMemset(dev_Selected_index, -1, sizeArray[6] * sizeof(int));
    checkErrorCuda(cudaStatus, "hipMemset failed! dev_Selected_index");
    cudaStatus = hipMemcpy(dev_E, E, sizeArray[7] * sizeof(double), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_E");

    double* extended_bestSpinModel = new double[sizeArray[9] * sizeArray[8]];
    for(int i=0;i< sizeArray[9];i++)
        memcpy(extended_bestSpinModel + i * sizeArray[8], bestSpinModel, sizeof(int) * sizeArray[8]);
    cudaStatus = hipMemcpy(dev_bestSpinModel, extended_bestSpinModel, sizeArray[9] * sizeArray[8] * sizeof(int), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_bestSpinModel");

    double* bestEnergyArray = new double[sizeArray[9]];
    fill1Darray(bestEnergyArray, bestEnergy, sizeArray[9]);
    cudaStatus = hipMemcpy(dev_bestenergy, bestEnergyArray, sizeArray[9] * sizeof(double), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_bestenergy");

    cudaStatus = hipMemcpy(dev_Temprature, Temprature, sizeArray[10] * sizeof(double), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_Temprature");

    //hipMemset(dev_bestenergy, bestEnergy, sizeof(double));
    



    //cudaStatus = hipMemcpy(dev_DelH_sign, DelH_sign, lenY * sizeof(int), hipMemcpyHostToDevice);
    //checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_DelH_sign");
    return cudaStatus;


}


hipError_t copyMemoryFromHostToDevice(double* H, double* dev_H, double* DelHGpu, double* dev_DelH, int* DelH_sign, int* dev_DelH_sign, double* WGpu, double* dev_W, double* B, double* dev_B, int* Y, int* dev_Y, int lenY, double* E, double* dev_E, int step, double bestEnergy, double* dev_bestenergy, int* bestSpinModel, int* dev_bestSpinModel, int replica, int block_size, int* dev_Selected_index) {
                    
    hipError_t cudaStatus;
    // Copy input vectors from host memory to GPU buffers.

    cudaStatus = hipMemcpy(dev_H, H, lenY * sizeof(double), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_H");
    cudaStatus = hipMemcpy(dev_DelH, DelHGpu, lenY * lenY * sizeof(double), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_DelH");
    cudaStatus = hipMemcpy(dev_W, WGpu, lenY * lenY * sizeof(double), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_W");
    //hipMemcpy(dev_B, B, lenY * sizeof(double), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_Y, Y, lenY * sizeof(int), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_Y");
    cudaStatus = hipMemcpy(dev_bestSpinModel, bestSpinModel, lenY * sizeof(int), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_bestSpinModel");
    cudaStatus = hipMemcpy(dev_bestenergy, &bestEnergy, sizeof(double), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_bestenergy");
    //hipMemset(dev_bestenergy, bestEnergy, sizeof(double));
    cudaStatus = hipMemcpy(dev_E, E + step - 1, sizeof(double), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_E");
    //cudaStatus = hipMemset(dev_lenY, lenY, sizeof(int));
    //checkErrorCuda(cudaStatus, "hipMemset failed! dev_lenY");
    cudaStatus = hipMemset(dev_Selected_index, -1, block_size * sizeof(int));
    checkErrorCuda(cudaStatus, "hipMemset failed! dev_Selected_index");


    cudaStatus = hipMemcpy(dev_DelH_sign, DelH_sign, lenY * sizeof(int), hipMemcpyHostToDevice);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_DelH_sign");
    return cudaStatus;


}

void copyMemoryFromDeviceToHost_with_size(int* vector_Y, int* dev_Y, int* bestSpinModel, int* dev_bestSpinModel, int& bestEnergy, double* dev_bestenergy, double* vector_E, double* dev_E, int* memory_size)
{
    hipError_t cudaStatus;

    //return cudaStatus;
}


hipError_t copyMemoryFromDeviceToHost(int lenY, double* H, double* dev_H, double* DelHGpu, double* dev_DelH, int* Y, int* dev_Y, int* bestSpinModel, int* dev_bestSpinModel, double& bestEnergy, double* dev_bestenergy, double* E, double* dev_E, int* DelH_sign, int* dev_DelH_sign, int step, int exchange_attempts) {
    hipError_t cudaStatus;

    cudaStatus = hipMemcpy(&bestEnergy, dev_bestenergy, sizeof(double), hipMemcpyDeviceToHost);
    printf("Value of bestEnergy: %lf\n", bestEnergy);
    
    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(H, dev_H, lenY * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(DelHGpu, dev_DelH, lenY * lenY * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Y, dev_Y, lenY * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(bestSpinModel, dev_bestSpinModel, lenY * sizeof(int), hipMemcpyDeviceToHost);
    
    hipMemcpy(E + step, dev_E, exchange_attempts * sizeof(double), hipMemcpyDeviceToHost);


    cudaStatus = hipMemcpy(DelH_sign, dev_DelH_sign, lenY * sizeof(int), hipMemcpyDeviceToHost);
    checkErrorCuda(cudaStatus, "hipMemcpy failed! dev_DelH_sign -> DelH_sign");

    

    return cudaStatus;
}

void FreeMemoryDevice(double* dev_H, double* dev_DelH, double* dev_W, double* dev_B, double* dev_E, int* dev_bestSpinModel, int* dev_Y, int* dev_Selected_index, int* dev_lenY, int* dev_DelH_sign, double* dev_bestenergy) {
    hipFree(dev_H);
    hipFree(dev_DelH);
    hipFree(dev_DelH_sign);
    hipFree(dev_W);
    hipFree(dev_B);
    hipFree(dev_Y);
    hipFree(dev_E);
    hipFree(dev_Selected_index);
    hipFree(dev_bestSpinModel);
    hipFree(dev_bestenergy);
}


//prepare_full_MetropolisKernel(vector_H, vector_DelH, DelH_sign, WGpu, B, vector_Y, lenY, vector_E, Temperature, exchange_attempts, bestEnergy, bestSpinModel);
// prepare memory to call Gpu Kernel
hipError_t prepare_full_MetropolisKernel(double* vector_H, double* vector_DelH, int* vector_DelH_sign, double* WGpu, double* B, int* vector_Y, int lenY, double* vector_E, double* Temperature, int exchange_attempts, double& bestEnergy, int* bestSpinModel, int replica, int numberOfIteration) {

    double* dev_H = 0;
    double* dev_DelH = 0;
    double* dev_W = 0;
    double* dev_B = 0;
    double* dev_E = 0;

    int* dev_bestSpinModel = 0;
    int* dev_Y = 0;
    int* dev_Selected_index = 0;
    int* dev_lenY = 0;
    int* dev_DelH_sign = 0;
    double* dev_bestenergy = 0;
    double* dev_Temprature = 0;

    

    int select_index_size = nextPowerOf2(lenY);

    int memory_size[] = {
        replica*lenY,                   //  0: dev_H
        replica* lenY*lenY,             //  1: dev_DelH
        replica,                        //  2: dev_DelH_sign
        lenY*lenY,                      //  3: dev_W
        lenY,                           //  4: dev_B
        replica* lenY,                  //  5: dev_Y
        replica* select_index_size,            //  6: dev_selected_index
        replica* numberOfIteration,     //  7: dev_E
        lenY,                           //  8: dev_bestSpinModel, actaual size of dev_bestSpinModel is replica * lenY
        replica,                        //  9: dev_bestenergy
        replica,                        // 10: dev_Temprature
        1                               // 11: dev_lenY
    };

    hipError_t cudaStatus;
    // Allocate GPU buffers for inputs and outputs.
    cudaStatus = allocateMemory_with_size(&dev_H, &dev_DelH, &dev_W, &dev_B, &dev_E, &dev_bestSpinModel, &dev_Y, &dev_Selected_index, &dev_lenY, &dev_DelH_sign, &dev_bestenergy, &dev_Temprature, memory_size);

    cudaStatus = copyMemoryFromHostToDevice_with_size(vector_H, dev_H, vector_DelH, dev_DelH, vector_DelH_sign, dev_DelH_sign, WGpu, dev_W, vector_Y, dev_Y, vector_E, dev_E, bestEnergy, dev_bestenergy, bestSpinModel, dev_bestSpinModel, dev_Selected_index, dev_Temprature, Temperature, memory_size);
    cout << "prepare_full_MetropolisKernel, after memory copy"<< endl;
    // Launch a kernel on the GPU with one thread for each element.
    printX(vector_Y + 3 * lenY, lenY, "X for replica 3:");
    printH(vector_H + 3*lenY, lenY, "H[3]: ");
    printH(vector_DelH + 3 * lenY * lenY, lenY * lenY, "DelH for replica 3: ");

    full_mode_metropolisKernel << <replica, lenY >> > (dev_H, dev_DelH, dev_DelH_sign, dev_Y, dev_Selected_index, select_index_size, dev_E, dev_bestSpinModel, dev_bestenergy, numberOfIteration, exchange_attempts, dev_Temprature);
    
    cout << "after full_mode_metropolisKernel" << endl;
    
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    checkErrorCuda(cudaStatus, "prepareMetropolisKernel launch failed : !");

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching metropolisKernel!\n", cudaStatus);
        cout << hipGetErrorString(cudaStatus) << endl;;
        goto Error;
    }



    // Copy output vector from GPU buffer to host memory.

    //copyMemoryFromDeviceToHost_with_size(vector_Y, dev_Y, bestSpinModel, dev_bestSpinModel, bestEnergy, dev_bestenergy, vector_E, dev_E, memory_size);

    //cout << " \t\t\t Best Energy: " << bestEnergy << endl;

Error:
    FreeMemoryDevice(dev_H, dev_DelH, dev_W, dev_B, dev_E, dev_bestSpinModel, dev_Y, dev_Selected_index, dev_lenY, dev_DelH_sign, dev_bestenergy);

    return cudaStatus;
}



// prepare memory to call Gpu Kernel
hipError_t prepareMetropolisKernel(double* H, double* DelHGpu, int* DelH_sign, double* WGpu, double* B, int* Y, int lenY, double* M, double* E, double T, int step, int exchange_attempts, double& bestEnergy, int* bestSpinModel, int replica) {
    
    double* dev_H = 0;
    double* dev_DelH = 0;
    double* dev_W = 0;
    double* dev_B = 0;
    double* dev_E = 0;

    int* dev_bestSpinModel = 0;
    int* dev_Y = 0;
    int* dev_Selected_index=0;
    int* dev_lenY = 0;
    int* dev_DelH_sign = 0;
    double* dev_bestenergy = 0;

    
    int block_size = nextPowerOf2(lenY);
    
    
        
    hipError_t cudaStatus;
    // Allocate GPU buffers for inputs and outputs.
    cudaStatus = allocateMemory(lenY, block_size, exchange_attempts, &dev_H, &dev_DelH, &dev_W, &dev_B, &dev_E, &dev_bestSpinModel, &dev_Y, &dev_Selected_index, &dev_lenY, &dev_DelH_sign, &dev_bestenergy);

    cudaStatus = copyMemoryFromHostToDevice(H, dev_H, DelHGpu, dev_DelH, DelH_sign, dev_DelH_sign, WGpu, dev_W, B, dev_B, Y, dev_Y, lenY, E, dev_E, step, bestEnergy, dev_bestenergy, bestSpinModel, dev_bestSpinModel, replica, block_size, dev_Selected_index);

    // Launch a kernel on the GPU with one thread for each element.
    metropolisKernel << <1, block_size >> > (dev_H, dev_DelH, dev_DelH_sign, dev_Y, dev_Selected_index, lenY, dev_E, dev_bestSpinModel, dev_bestenergy, exchange_attempts, T);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    checkErrorCuda(cudaStatus, "prepareMetropolisKernel launch failed : !");

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching metropolisKernel!\n", cudaStatus);
        cout << hipGetErrorString(cudaStatus) << endl;;
        goto Error;
    }


    
    // Copy output vector from GPU buffer to host memory.

    copyMemoryFromDeviceToHost(lenY, H, dev_H, DelHGpu, dev_DelH, Y, dev_Y, bestSpinModel, dev_bestSpinModel, bestEnergy, dev_bestenergy, E, dev_E, DelH_sign, dev_DelH_sign, step, exchange_attempts);
    
    cout << " \t\t\t Best Energy: " << bestEnergy << endl;

Error:
    FreeMemoryDevice(dev_H, dev_DelH, dev_W, dev_B, dev_E, dev_bestSpinModel, dev_Y, dev_Selected_index, dev_lenY, dev_DelH_sign, dev_bestenergy);

    return cudaStatus;
}


/* ***************************************************************** */




// One step in metropolis algorithm
double metropolis(int ExecuteMode, double** W, double* B, double* H, double** DelH, int* X, int lenX, double OldE, double T, int step, int replica) {
    double E = 0;
    int i = rand() % lenX;

    double deltaE;
    if (ExecuteMode == IsingMode)    deltaE = deltaEnergyIsing(ExecuteMode, W, B, X, lenX, i);
    else if (ExecuteMode == QUBOMode) deltaE = deltaEnergyQUBO(W, X, i, H);

    if ((deltaE < 0) || ((rand() / static_cast<double>(RAND_MAX)) < exp(-deltaE / T))) {
        if (ExecuteMode == IsingMode) {
            X[i] *= -1;
            E = energy(W, B, X, lenX);
        }
        else if (ExecuteMode == QUBOMode) {
            X[i] = 1 - X[i];
            E = Energy_based_Delta(OldE, deltaE);
            updateH(H, DelH, lenX, i);
            update_delta_H(DelH, i, lenX);
        }
        return E;
    }
    return OldE;
}

//initialize the bestEnergy, Temperature array range, Energy (E), Magnet (M), and bestSpinModel

double initEnergyAndMagnet(int num_replicas, double** W, double* B, int** Y, int lenY, double** M, double** E, int* bestSpinModel){
    double bestEnergy = 0;
    if (num_replicas != 1) {
        for (int r = 0; r < num_replicas; r++) {
            E[r][0] = energy(W, B, Y[r], lenY);

            M[r][0] = magnetization(Y[r], lenY);
            if (r == 0 || bestEnergy > E[r][0]) {
                bestEnergy = E[r][0];
                memcpy(bestSpinModel, Y[r], sizeof(int) * lenY);
            }
        }
    }
    else {
        E[0][0] = energy(W, B, Y[0], lenY);
        M[0][0] = magnetization(Y[0], lenY);
        bestEnergy = E[0][0];
        memcpy(bestSpinModel, Y[0], sizeof(int) * lenY);
    }
    return bestEnergy;
}


void intitTemperature(int num_replicas, double minTemp, double maxTemp, double* Temperature){
    if (num_replicas != 1) {
        for (int r = 0; r < num_replicas; r++) {
            Temperature[r] = minTemp + r * (maxTemp - minTemp) / (num_replicas - 1);
            cout << "Temperature: " << Temperature[r] << endl;
        }
    }
    else {
        Temperature[0] = minTemp;
        cout << "Temperature: " << Temperature[0] << endl;
    }
}


double full_GPU_Mode(int num_replicas, double** W, double* B, int** Y, int lenY, double** M, double** E, int* bestSpinModel, int numberOfIteration, int exchange_attempts, double minTemp, double maxTemp) {
    double bestEnergy;
    //double* Temperature = new double[num_replicas];
    double** H = ComputeH_forAllReplica(num_replicas, W, B, Y, lenY);
    double*** DelH = ComputeDelH_forAllReplica(num_replicas, W, Y, lenY);
    int* DelH_sign = new int[num_replicas* lenY];
    //fill2DarrayInt(DelH_sign, 1, num_replicas, lenY);
    double* vector_DelH;
    double* vector_H;
    double* vector_E;
    int* vector_Y;
    double* vector_W = convert2Dto1D(W, lenY, lenY);;

    double* Temperature = new double[num_replicas];
    intitTemperature(num_replicas, minTemp, maxTemp, Temperature);
    bestEnergy = initEnergyAndMagnet(num_replicas, W, B, Y, lenY, M, E, bestSpinModel);
    cout << bestEnergy << endl;

    vector_DelH = VectorizedDelH(DelH, num_replicas, lenY);
    vector_H = convert2Dto1D(H, num_replicas, lenY);
    //printAllH(H, num_replicas, lenY);
    //printH(vector_H, num_replicas * lenY, "vector_H");
    vector_Y = convert_int_2Dto1D(Y, num_replicas, lenY);
    vector_E = convert2Dto1D(E, num_replicas, numberOfIteration);

    // Preperation of replica exchange parameters
    //int exchangeFlag = 0;   // Flag to enable the exchange between neighbour replicas
    // Perform the Metropolis function numberOfIteration times for each replica 
    //for (int step = 1; step < numberOfIteration; step++) {
        //cout << "step: " << step << endl;
        //cout << "******************* before prepareMetropolisKernel calling/ step: " << step << endl;
        
        prepare_full_MetropolisKernel(vector_H, vector_DelH, DelH_sign, vector_W, B, vector_Y, lenY, vector_E, Temperature, exchange_attempts, bestEnergy, bestSpinModel, num_replicas, numberOfIteration);

        
      
        // Replica exchange attempts
        
        //step = step + exchange_attempts - 1;
        //replicaExchangeGpu(Temperature, num_replicas, Y, E, step, H, DelHGpu, DelH_sign, exchangeFlag);

        //cout << "after prepareMetropolisKernel calling/ step: " << step << endl;
        

    //}

    return bestEnergy;
}

/* Execute the optimization function based on replica exchange MCMC
* ExecuteMode : Ising or QUBO
* W: interconnection of spins
* B: bias
* Y: spins 2D array sizeof (num_replicas \times lenY), lenY: number of spins
* M: record of Magnet in each iteration per replica
* E: record of Energy in each iteration per replica
*/
void ising(int ExecuteMode, double** W, double* B, int** Y, int lenY, double** M, double** E, double T, int num_replicas, int numberOfIteration, int exchange_attempts, int* bestSpinModel, double minTemp, double maxTemp) {

    double bestEnergy;
    double* Temperature = new double[num_replicas];
    double** H = ComputeH_forAllReplica(num_replicas, W, B, Y, lenY);
    double*** DelH = ComputeDelH_forAllReplica(num_replicas, W, Y, lenY);
    int** DelH_sign = Declare2D_ArrayInt(num_replicas, lenY);
    fill2DarrayInt(DelH_sign, 1, num_replicas, lenY);
    double** DelHGpu;
    double* WGpu;
    
    

    
    //initialize the bestEnergy, Temperature array range, Energy (E), Magnet (M), and bestSpinModel
    /*
    if (num_replicas != 1) {
        for (int r = 0; r < num_replicas; r++) {
            Temperature[r] = minTemp + r * (maxTemp - minTemp) / (num_replicas - 1);
            cout << "Temperature: " << Temperature[r] << endl;
            E[r][0] = energy(W, B, Y[r], lenY);

            M[r][0] = magnetization(Y[r], lenY);
            if (r == 0 || bestEnergy > E[r][0]) {
                bestEnergy = E[r][0];
                memcpy(bestSpinModel, Y[r], sizeof(int) * lenY);
            }
        }
    }
    else {
        Temperature[0] = minTemp;
        cout << "Temperature: " << Temperature[0] << endl;
        E[0][0] = energy(W, B, Y[0], lenY);
        M[0][0] = magnetization(Y[0], lenY);
        bestEnergy = E[0][0];
        memcpy(bestSpinModel, Y[0], sizeof(int) * lenY);
    }

    cout << bestEnergy<< endl;
    */
    intitTemperature(num_replicas, minTemp, maxTemp, Temperature);
    bestEnergy = initEnergyAndMagnet(num_replicas, W, B, Y, lenY, M, E, bestSpinModel);

    cout << bestEnergy << endl;

    // Vectorization of parammeters for Gpu
    if (ExecuteMode == QUBOGPU) {
        DelHGpu = convertDelHtoGpuDelH(DelH, num_replicas, lenY);
        WGpu = convert2Dto1D(W, lenY, lenY);
        
    }
    else if (ExecuteMode == QUBOGPUFULL) {
        //prepare_Full_GPU_Mode();
        full_GPU_Mode(num_replicas, W, B, Y, lenY, M, E, bestSpinModel, numberOfIteration, exchange_attempts, minTemp, maxTemp);
        return;
    }    

    // Preperation of replica exchange parameters
    int exchangeFlag = 0;   // Flag to enable the exchange between neighbour replicas
    // Perform the Metropolis function numberOfIteration times for each replica 
    for (int step = 1; step < numberOfIteration; step++) {
        //cout << "step: " << step << endl;
        //cout << "******************* before prepareMetropolisKernel calling/ step: " << step << endl;
        for (int r = 0; r < num_replicas; r++) {
            T = Temperature[r];
            double previousE = E[r][step - 1];
            //for (int spin = 0; spin < lenY/25; spin++) 
            
            if (ExecuteMode == QUBOGPU) {                
                //printX(Y[r], lenY, "Y["+to_string(r)+"]");
                //printH(H[r], lenY, "H");
                //print2D_arr_double(DelH[r], lenY, lenY);
                prepareMetropolisKernel(H[r], DelHGpu[r], DelH_sign[r], WGpu, B, Y[r], lenY, M[r], E[r], T, step, exchange_attempts, bestEnergy, bestSpinModel, r);
                    
            }
            else {
                previousE = metropolis(ExecuteMode, W, B, H[r], DelH[r], Y[r], lenY, previousE, T, step, r);
                E[r][step] = previousE;
                M[r][step] = magnetization(Y[r], lenY);
                if (bestEnergy > E[r][step]) {
                    memcpy(bestSpinModel, Y[r], sizeof(int) * lenY);
                    bestEnergy = E[r][step];
                }
            }            
        }
        // Replica exchange attempts
        if (ExecuteMode != QUBOGPU && step % exchange_attempts == 0) {
            replicaExchange(Temperature, num_replicas, Y, M, E, step, H, DelH, exchangeFlag);
        }
        else if (ExecuteMode == QUBOGPU) {
            step = step + exchange_attempts-1;
            replicaExchangeGpu(Temperature, num_replicas, Y, E, step, H, DelHGpu, DelH_sign, exchangeFlag);
            
            cout << "after prepareMetropolisKernel calling/ step: " << step << endl;
        }

    }
}

/**********************************************************************************************/





int main()
{
    


    int L = 0;                      // Number of spins for each replica
    int Lsqrt = 0;                  /* This parameter is just used for graphical representation of 2-D Ising model
                                     and it is not important for other cases Lsqrt * Lsqrt = L         */
    double T = 0.0;                 // Temperature ---> when 
    int num_replicas = 1;           // Number of replica in the replica exchange MCMC
    double minTemp = 0;             // Min temperature of replica exchange MCMC
    double maxTemp = 0;             // Max temperature of replica exchange MCMC
    int numberOfIteration = 1;      // Number of Iteration in for MCMC
    int exchange_attempts = 0;      // After how many iteration, an exchange should be applied

    /*      E = -\sum_{i,j} A_{i,j} s_i s_j - \sum_i B_i s_i   */
    string Afile = "";              //      file path for A matrix in Ising (QUBO) model 
    string Bfile = "";              //      file path for Bias (B) in Ising (QUBO) model 
    string outputPath = "";          //     The path to the directory to save the output 
    int ExecuteMode = QUBOMode;     //      Execution mode: IsingMode or QUBOMode

    /*      Read the setting file and initialize the parameters    */
    readSetting(L, Lsqrt, Afile, Bfile, outputPath, ExecuteMode, num_replicas, numberOfIteration, exchange_attempts, minTemp, maxTemp);

    //cout << "L: " << L << " Lsqrt: " << Lsqrt << " Afile: " << Afile << " Bfile: " << Bfile << " ExecuteMode: " << ExecuteMode << " num_replicas: " << num_replicas << " numberOfIteration: " << numberOfIteration << " exchange_attempts: " << exchange_attempts << endl;
    //cout << "minTemp: " << minTemp << " maxTemp: " << maxTemp << endl;


    // Create the output folder
    createFolder(outputPath);

    //Initialize the spins for each replica
    int** X = createVector(ExecuteMode, L, num_replicas);
    int* bestSpinModel = new int[L];            // Best found solution

    double** M = new double* [num_replicas];     //  Magnetization for different replica (This parameter is usefull for Ising model)
    double** E = new double* [num_replicas];     //  Energy for different replica
    for (int r = 0; r < num_replicas; r++) {
        M[r] = new double[numberOfIteration];
        E[r] = new double[numberOfIteration];
    }

    //  Initialize the bias    
    double* B;
    B = initB(L, ReadDataFromFile, Bfile);

    //Initalize the interconnection of spins
    double** A;
    A = initW(L, Lsqrt, ReadDataFromFile, Afile);
    //writeMatrixToFile(outputPath + "\\WInit.csv", A, L);

    // Log all the initial spin states
    writeSpinsInFile(num_replicas, X, L, Lsqrt, outputPath, "Initlattice");

    // Optimization Function
    ising(ExecuteMode, A, B, X, L, M, E, T, num_replicas, numberOfIteration, exchange_attempts, bestSpinModel, minTemp, maxTemp);

    //  Record Logs: Magnet, Energy, final spin states, and best spin model
    recordLogs(outputPath, M, E, numberOfIteration, num_replicas, L, Lsqrt, X, bestSpinModel);

    return 0;
}



/*****************************************************************************************/